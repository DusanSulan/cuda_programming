// This program computes the sum of two vectors of length N
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <array>
#include <algorithm>
#include <iterator>

using std::begin;
using std::end;
using std::copy;
using std::generate;
using std::array;
using std::cout;
using std::endl;

// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(int* a, int* b, int* c, int N) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  // Boundary check
  if (tid < N) {
    // Each thread adds a single element
    c[tid] = a[tid] + b[tid];
  }
}

// Check vector add result
// Templatize this to handle multiple array inputs
template <size_t SIZE>
void verify_result(array<int, SIZE>a, array<int, SIZE> b, array<int, SIZE> c) {
  for(int i = 0; i < SIZE; i++){
    assert(c[i] == a[i] + b[i]);
  }
}

int main() {
  // Vector size of 2^16 (65536 elements)
  const int N = 1 << 16;
  size_t bytes = sizeof(int) * N;

  // Arrays for holding the host-side (cpu-side) data
  array<int, N> a;
  array<int, N> b;
  array<int, N> c;

  // Initialize random numbers in each array
  generate(begin(a), end(a), [] () { return rand() % 100; });
  generate(begin(b), end(b), [] () { return rand() % 100; });

  // Allocate memory on the device
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data from the host to the device (cpu -> gpu)
  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA (1024 threads per CTA)
  int NUM_THREADS = 1 << 10;

  // CTAs per Grid
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  // Launch the kernel on the GPU
  // Kernel calls are asynchronous (the CPU program continues execution after
  // call, but no necessarily before the kernel finishes)
  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

  // Copy sum vector from device to host
  // cudaMemcpy is a synchronous operation, and waits for the prior kernel
  // launch to complete (both go to the default stream in this case).
  // Therefore, this cudaMemcpy acts as both a memcpy and synchronization
  // barrier.
  hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  // Check result for errors
  verify_result<a.size()>(a, b, c);

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  printf("COMPLETED SUCCESFULLY\n");

  return 0;
}